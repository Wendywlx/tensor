﻿#include <matazure/tensor>

using namespace matazure;

int main() {
	cuda::matrix<float, row_major_layout> cmat_lhs(17, 38);
	fill(cmat_lhs, 1.0f);
	cuda::matrix<float> cmat_rhs(38, 19);
	fill(cmat_rhs, 2.0f);

	//通用乘法操作
	auto cmat_re = puzzle::prod_general(cmat_lhs, cmat_rhs).persist();
	cuda::device_synchronize();

	//cuda的分块矩阵乘法
	auto cmat_re_block = cuda::puzzle::prod_block<16>(cmat_lhs, cmat_rhs);
	cuda::device_synchronize();

	auto mat_re = mem_clone(cmat_re, host_tag{});
	auto mat_re_block = mem_clone(cmat_re_block, host_tag{});

	//验证结果一致
	for (int_t j = 0; j < mat_re.shape()[1]; ++j) {
		for (int_t i = 0; i < mat_re.shape()[0]; ++i) {
			MATAZURE_ASSERT(mat_re(i, j) == mat_re_block(i, j), "failed");
		}
	}

	return 0;
}
